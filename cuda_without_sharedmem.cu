#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "/content/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "/content/stb_image_write.h"

#define MAX_PATH 255
#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32
#define r 5

__global__ void blur(uint8_t *input_img, uint8_t *output_img, int width,
                     int height, int channels) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= width || y >= height) return;
    int i_img = (y * width + x) * channels;
    int count = 0;
    int output_red = 0, output_green = 0, output_blue = 0;
    for (int x_box = x - r; x_box < x + r + 1; x_box++) {
        for (int y_box = y - r; y_box < y + r + 1;
             y_box++) {
            if (x_box >= 0 && x_box < width && y_box >= 0 && y_box < height) {
                int i_box = (y_box * width + x_box) * channels;
                output_red += input_img[i_box];
                output_green += input_img[i_box + 1];
                output_blue += input_img[i_box + 2];
                count++;
            }
        }
    }
    output_img[i_img] = output_red / count;
    output_img[i_img + 1] = output_green / count;
    output_img[i_img + 2] = output_blue / count;
    if (channels == 4) output_img[i_img + 3] = input_img[i_img + 3];
}

const char *get_file_ext(char *file_path) {
    const char *p, *dot = file_path;
    while (p = strchr(dot, '.')) dot = p + 1;
    if (dot == file_path) return "";
    return dot;
}

int main(int argc, char **argv) {
    char input_file[MAX_PATH + 1], output_file[MAX_PATH + 1];
    const char *input_file_extension;

    strncpy(input_file, "/content/320x240.jpg",MAX_PATH);
    input_file[MAX_PATH] = '\0';
    input_file_extension = get_file_ext(input_file);
    strncpy(output_file, "/content/photo2_blurr.jpg", MAX_PATH);
    output_file[MAX_PATH] = '\0';

    int width, height, channels;
    if (stbi_info(input_file, &width, &height, &channels) && channels != 4 &&
        channels != 3) {
        printf("Invalid input image '%s' has %d channel%s, expected 3 or 4\n",
               input_file, channels, channels > 1 ? "s" : "");
        exit(1);
    }
    uint8_t *input_img = stbi_load(input_file, &width, &height, &channels, 0);
    if (!input_img) {
        printf("Error in loading the image\n");
        exit(1);
    }
    printf(
        "Loaded image '%s' with a width of %dpx, a height of %dpx and %d "
        "channels\n",
        input_file, width, height, channels);
    size_t img_size = width * height * channels;
    uint8_t *output_img = (uint8_t *)malloc(img_size);
    if (!output_img) {
        printf("Unable to allocate memory for the output image\n");
        exit(1);
    }
    uint8_t *d_input_img, *d_output_img;
    hipEvent_t start, stop;
    float time_spent;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void **)&d_input_img, img_size);
    hipMalloc((void **)&d_output_img, img_size);


    hipMemcpy(d_input_img, input_img, img_size, hipMemcpyHostToDevice);


    const dim3 block_size(BLOCK_WIDTH, BLOCK_HEIGHT, 1);
    unsigned int nb_blocksx = (unsigned int)(width / BLOCK_WIDTH + 1);
    unsigned int nb_blocksy = (unsigned int)(height / BLOCK_HEIGHT + 1);
    const dim3 grid_size(nb_blocksx, nb_blocksy, 1);

    blur<<<grid_size, block_size>>>(d_input_img, d_output_img, width, height,
                                    channels);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Cuda error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_spent, start, stop);
    hipMemcpy(output_img, d_output_img, img_size, hipMemcpyDeviceToHost);
    const char *output_file_extension = get_file_ext(output_file);
    if ((strcmp(output_file_extension, "jpg") ||
          strcmp(output_file_extension, "jpeg") ||
          strcmp(output_file_extension, "JPG") ||
          strcmp(output_file_extension, "JPEG")))
        stbi_write_jpg(output_file, width, height, channels, output_img, 100);
    else if (!(strcmp(output_file_extension, "bmp") ||
               strcmp(output_file_extension, "BMP")))
        stbi_write_bmp(output_file, width, height, channels, output_img);
    else
        stbi_write_png(output_file, width, height, channels, output_img,
                       width * channels);
    stbi_image_free(input_img);
    free(output_img);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_input_img);
    hipFree(d_output_img);
    printf(
        "Check '%s' (took %fms with (%d, %d) block dim and (%d, %d) grid "
        "dim)\n",
        output_file, time_spent, BLOCK_WIDTH, BLOCK_HEIGHT, nb_blocksx,
        nb_blocksy);
    return 0;
}
